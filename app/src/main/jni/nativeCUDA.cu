#include "hip/hip_runtime.h"
#include "nativeCUDA.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define DEBUG
inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			LOGI("CUDA Runtime Error: %sn", hipGetErrorString(result));
			//assert(result == hipSuccess);
		}
#endif
		return result;
}

__global__ void addKernel(float* d_a, float* d_b, float* d_ret, int n);

void launchAddKernel(float* d_a, float* d_b, float* d_ret, int n) {
    addKernel<<<(n + TPB-1) / TPB, TPB>>>(d_a, d_b, d_ret, n);
}

float* CUDA_addVectors(float* a, float* b, int n) {
    size_t arr_size = n * sizeof(float);

    // Allocate space for sum
    float *ret, *d_ret;
    checkCuda( hipHostMalloc((void**) &ret, arr_size) ); // Host
    checkCuda( hipMalloc((void**) &d_ret, arr_size) ); // Device
    // Allocate device space for a and b
    float *d_a, *d_b;
    checkCuda (hipMalloc((void**) &d_a, arr_size) );
    checkCuda (hipMalloc((void**) &d_b, arr_size) );
    // Copy a and b to device memory asynchronously
    checkCuda( hipMemcpyAsync(d_a, a, arr_size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyAsync(d_b, b, arr_size, hipMemcpyHostToDevice) );
    // Wait for copies to complete
    hipDeviceSynchronize();

    // Launch device kernel
    launchAddKernel(d_a, d_b, d_ret, n);
    // Wait for kernel to finish
    hipDeviceSynchronize();
    // Check for any errors created by kernel
    checkCuda(hipGetLastError());

    // Copy back sum array
    checkCuda( hipMemcpy(ret, d_ret, arr_size, hipMemcpyDeviceToHost) );

    // Free allocated memory
    hipFree(d_ret);
    hipFree(d_a);
    hipFree(d_b);

    return ret;
}

// GPU kernel
__global__ void addKernel(float* d_a, float* d_b, float* d_ret, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= n) {
        return;
    }
    d_ret[index] = d_a[index] + d_b[index];
}